#include "hip/hip_runtime.h"
#include "tensor_aclr_impl.cuh"
#include <iostream>
#include <chrono>

using namespace std;
int main() {
    ts::acceleration= true;
    ts::Tensor t = ts::tensor({3,3},
                              new double[9]{
                                      1,2,3,
                                      4,5,6,
                                      7,8,9 });
    ts::Tensor t1 = ts::tensor({3,3},
                              new double[9]{
                                      2,3,
                                      4,5,6,
                                      7,8,9,10 });
    auto start = std::chrono::high_resolution_clock::now();

    std::cout << (t-t1)<< std::endl;
    auto end = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> diff = end-start;
    std::cout << "Time to run the code: " << diff.count() << " s\n";

//    std::cout << ts::einsum("iii",{t})<< std::endl;

//    std::cout << t.transpose(0,1) << std::endl;
//    std::cout << t.view({5,3})<<std::endl;

//    std::cout << t << std::endl;
//    std::cout << t.sum(0) << std::endl;
//    std::cout << sum(t,1) << std::endl;
//    std::cout << max(t,1) << std::endl;


    return 0;
}
